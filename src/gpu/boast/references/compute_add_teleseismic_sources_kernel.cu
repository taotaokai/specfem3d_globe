
#include <hip/hip_runtime.h>
#define NDIM 3
#define NGLLX 5
#define NGLL2 25
#define INDEX2(xsize,x,y) x + (y)*xsize
#define INDEX3(xsize,ysize,x,y,z) x + xsize*(y + ysize*z)
#define INDEX4(xsize,ysize,zsize,x,y,z,i) x + xsize*(y + ysize*(z + zsize*i))

typedef float realw;

__global__ void compute_add_teleseismic_sources_kernel(realw* accel,
                                                       realw* field_teleseismic,
                                                       int interface_type,
                                                       int num_teleseismic_boundary_faces,
                                                       int* teleseismic_boundary_ispec,
                                                       int* ibool) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,k,iglob,ispec;

  // don't compute surface faces outside of range
  // and don't compute points outside NGLLSQUARE==NGLL2==25
  //if(igll < NGLL2 && iface < num_teleseismic_boundary_faces) {

  // way 2: only check face, no further check needed since blocksize = 25
  if (iface < num_teleseismic_boundary_faces){

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = teleseismic_boundary_ispec[iface]-1;

    // determines indices i,j,k depending on teleseismic boundary type
    switch( interface_type ){
      case 0:
        // xmin
        i = 0; // index -1
        k = (igll/NGLLX);
        j = (igll-k*NGLLX);
        break;
      case 1:
        // xmax
        i = NGLLX-1;
        k = (igll/NGLLX);
        j = (igll-k*NGLLX);
        break;
      case 2:
        // ymin
        j = 0;
        k = (igll/NGLLX);
        i = (igll-k*NGLLX);
        break;
      case 3:
        // ymax
        j = NGLLX-1;
        k = (igll/NGLLX);
        i = (igll-k*NGLLX);
        break;
      case 4:
        // zmin
        k = 0;
        j = (igll/NGLLX);
        i = (igll-j*NGLLX);
        break;
    }

    iglob = ibool[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

    atomicAdd(&accel[iglob*3  ], field_teleseismic[INDEX3(NDIM,NGLL2,0,igll,iface)]);
    atomicAdd(&accel[iglob*3+1], field_teleseismic[INDEX3(NDIM,NGLL2,1,igll,iface)]);
    atomicAdd(&accel[iglob*3+2], field_teleseismic[INDEX3(NDIM,NGLL2,2,igll,iface)]);

  } // num_teleseismic_boundary_faces
}

