#include "hip/hip_runtime.h"
#define NDIM 3
#define NGLLX 5
#define NGLL2 25
#define INDEX2(xsize,x,y) x + (y)*xsize
#define INDEX3(xsize,ysize,x,y,z) x + xsize*(y + ysize*z)
#define INDEX4(xsize,ysize,zsize,x,y,z,i) x + xsize*(y + ysize*(z + zsize*i))

typedef float realw;

__global__ void compute_teleseismic_gradient_kernel(realw* accel,
                                                    int interface_type,
                                                    int num_teleseismic_boundary_faces,
                                                    int* teleseismic_boundary_ispec,
                                                    realw* teleseismic_boundary_area,
                                                    int* ibool,
                                                    realw* d_field_teleseismic,
                                                    realw scale_displ) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,k,iglob,ispec;
  realw area;

  // don't compute surface faces outside of range
  // and don't compute points outside NGLLSQUARE==NGLL2==25
  //if(igll < NGLL2 && iface < num_teleseismic_boundary_faces) {

  // way 2: only check face, no further check needed since blocksize = 25
  if (iface < num_teleseismic_boundary_faces){

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = teleseismic_boundary_ispec[iface]-1;

    // determines indices i,j,k depending on teleseismic boundary type
    switch( interface_type ){
      case 0:
        // xmin
        i = 0; // index -1
        k = (igll/NGLLX);
        j = (igll-k*NGLLX);
        break;
      case 1:
        // xmax
        i = NGLLX-1;
        k = (igll/NGLLX);
        j = (igll-k*NGLLX);
        break;
      case 2:
        // ymin
        j = 0;
        k = (igll/NGLLX);
        i = (igll-k*NGLLX);
        break;
      case 3:
        // ymax
        j = NGLLX-1;
        k = (igll/NGLLX);
        i = (igll-k*NGLLX);
        break;
      case 4:
        // zmin
        k = 0;
        j = (igll/NGLLX);
        i = (igll-j*NGLLX);
        break;
    }

    iglob = ibool[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

    // gets associated area
    area = teleseismic_boundary_area[INDEX2(NGLL2,igll,iface)];

    d_field_teleseismic[INDEX3(NDIM,NGLL2,0,igll,iface)] = &
      accel[iglob*3]*scale_displ*area;

    d_field_teleseismic[INDEX3(NDIM,NGLL2,1,igll,iface)] = &
      accel[iglob*3+1]*scale_displ*area;

    d_field_teleseismic[INDEX3(NDIM,NGLL2,2,igll,iface)] = &
      accel[iglob*3+2]*scale_displ*area;

  } // num_teleseismic_boundary_faces
}

